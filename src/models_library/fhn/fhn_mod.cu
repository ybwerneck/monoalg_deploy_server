#include "hip/hip_runtime.h"
#include "../../gpu_utils/gpu_utils.h"
#include <stddef.h>
#include <stdint.h>

#include "fhn_mod.h"

extern "C" SET_ODE_INITIAL_CONDITIONS_GPU(set_model_initial_conditions_gpu) {

    log_info("Using modified FHN 1961 GPU model\n");

    uint32_t num_volumes = solver->original_num_cells;

    // execution configuration
    const int GRID  = (num_volumes + BLOCK_SIZE - 1)/BLOCK_SIZE;

    size_t size = num_volumes*sizeof(real);

    check_cuda_error(hipMallocPitch((void **) &(solver->sv), &pitch_h, size, (size_t )NEQ));
    check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(pitch), &pitch_h, sizeof(size_t)));

    real *initial_conditions = NULL;
    real *initial_conditions_device = NULL;

    if(solver->ode_extra_data) {
        initial_conditions = (real *)solver->ode_extra_data;
        check_cuda_error(hipMemcpy2D (solver->sv, pitch_h, initial_conditions, size, size, (size_t) NEQ, hipMemcpyHostToDevice));

    }
    else {
        kernel_set_model_inital_conditions <<<GRID, BLOCK_SIZE>>>(solver->sv, num_volumes);
    }

    check_cuda_error( hipPeekAtLastError() );
    hipDeviceSynchronize();

    check_cuda_error(hipFree(initial_conditions_device));

    return pitch_h;

}

extern "C" SOLVE_MODEL_ODES(solve_model_odes_gpu) {

    size_t num_cells_to_solve = ode_solver->num_cells_to_solve;
    uint32_t * cells_to_solve = ode_solver->cells_to_solve;
    real *sv = ode_solver->sv;
    real dt = ode_solver->min_dt;
    uint32_t num_steps = ode_solver->num_steps;


    // execution configuration
    const int GRID  = ((int)num_cells_to_solve + BLOCK_SIZE - 1)/BLOCK_SIZE;

    size_t stim_currents_size = sizeof(real)*num_cells_to_solve;
    size_t cells_to_solve_size = sizeof(uint32_t)*num_cells_to_solve;

    real *stims_currents_device;
    check_cuda_error(hipMalloc((void **) &stims_currents_device, stim_currents_size));
    check_cuda_error(hipMemcpy(stims_currents_device, stim_currents, stim_currents_size, hipMemcpyHostToDevice));


    //the array cells to solve is passed when we are using and adaptive mesh
    uint32_t *cells_to_solve_device = NULL;
    if(cells_to_solve != NULL) {
        check_cuda_error(hipMalloc((void **) &cells_to_solve_device, cells_to_solve_size));
        check_cuda_error(hipMemcpy(cells_to_solve_device, cells_to_solve, cells_to_solve_size, hipMemcpyHostToDevice));
    }
    solve_gpu <<<GRID, BLOCK_SIZE>>>(dt, sv, stims_currents_device, cells_to_solve_device, num_cells_to_solve, num_steps);

    check_cuda_error( hipPeekAtLastError() );

    check_cuda_error(hipFree(stims_currents_device));
    if(cells_to_solve_device) check_cuda_error(hipFree(cells_to_solve_device));

}

__global__ void kernel_set_model_inital_conditions(real *sv, int num_volumes) {
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;

    if (threadID < num_volumes) {

         *((real * )((char *) sv + pitch * 0) + threadID) = INITIAL_V; //u dimensionless
         *((real * )((char *) sv + pitch * 1) + threadID) = 0.0f; //v dimensionless

    }
}

// Solving the model for each cell in the tissue matrix ni x nj
__global__ void solve_gpu(real dt, real *sv, real* stim_currents,
                          uint32_t *cells_to_solve, uint32_t num_cells_to_solve,
                          int num_steps)
{
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int sv_id;

    // Each thread solves one cell model
    if(threadID < num_cells_to_solve) {
        if(cells_to_solve)
            sv_id = cells_to_solve[threadID];
        else
            sv_id = threadID;

        real rDY[NEQ];

        for (int n = 0; n < num_steps; ++n) {

            RHS_gpu(sv, rDY, stim_currents[threadID], sv_id);

            for(int i = 0; i < NEQ; i++) {
                *((real *) ((char *) sv + pitch * i) + sv_id) = dt * rDY[i] + *((real *) ((char *) sv + pitch * i) + sv_id);
            }            

        }

    }
}

inline __device__ void RHS_gpu(real *sv_, real *rDY_, real stim_current, int threadID_) {

    //State variables
    const real u = *((real*)((char*)sv_ + pitch * 0) + threadID_);
    const real v = *((real*)((char*)sv_ + pitch * 1) + threadID_);

    const real a = 0.2f;
    const real b = 0.5f;
    const real k = 36.0;
    const real epsilon  =  0.00040;



    rDY_[0] = k*(u*(1.0f - u)*(u - a) - u*v) + stim_current;
    rDY_[1] = k*epsilon*(b*u - v);


}

