#include "hip/hip_runtime.h"
#include "ToRORd_dynCl_mixed_endo_mid_epi.h"
#include <stddef.h>
#include <stdint.h>

__global__ void kernel_set_model_initial_conditions(real *sv, int num_volumes, size_t pitch, bool use_adpt_dt, real min_dt) {
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;

    if (threadID < num_volumes) {
        for (int i = 0; i < NEQ; i++) {
            // Initial conditions 200 beats (endocardium cell)
            *((real * )((char *) sv + pitch * 0) + threadID) = -9.035192e+01;
            *((real * )((char *) sv + pitch * 1) + threadID) = 1.162900e-02;
            *((real * )((char *) sv + pitch * 2) + threadID) = 6.500000e-05;
            *((real * )((char *) sv + pitch * 3) + threadID) = 1.239893e+01;
            *((real * )((char *) sv + pitch * 4) + threadID) = 1.239926e+01;
            *((real * )((char *) sv + pitch * 5) + threadID) = 1.482415e+02;
            *((real * )((char *) sv + pitch * 6) + threadID) = 1.482414e+02;
            *((real * )((char *) sv + pitch * 7) + threadID) = 1.527292e+00;
            *((real * )((char *) sv + pitch * 8) + threadID) = 1.524395e+00;
            *((real * )((char *) sv + pitch * 9) + threadID) = 7.400000e-05;
            *((real * )((char *) sv + pitch * 10) + threadID) = 5.720000e-04;
            *((real * )((char *) sv + pitch * 11) + threadID) = 8.579420e-01;
            *((real * )((char *) sv + pitch * 12) + threadID) = 8.577990e-01;
            *((real * )((char *) sv + pitch * 13) + threadID) = 7.199660e-01;
            *((real * )((char *) sv + pitch * 14) + threadID) = 8.575760e-01;
            *((real * )((char *) sv + pitch * 15) + threadID) = 1.200000e-04;
            *((real * )((char *) sv + pitch * 16) + threadID) = 5.748970e-01;
            *((real * )((char *) sv + pitch * 17) + threadID) = 3.250180e-01;
            *((real * )((char *) sv + pitch * 18) + threadID) = 8.540000e-04;
            *((real * )((char *) sv + pitch * 19) + threadID) = 9.997050e-01;
            *((real * )((char *) sv + pitch * 20) + threadID) = 5.959350e-01;
            *((real * )((char *) sv + pitch * 21) + threadID) = 4.350000e-04;
            *((real * )((char *) sv + pitch * 22) + threadID) = 9.997050e-01;
            *((real * )((char *) sv + pitch * 23) + threadID) = 6.589890e-01;
            *((real * )((char *) sv + pitch * 24) + threadID) = 0.000000e+00;
            *((real * )((char *) sv + pitch * 25) + threadID) = 1.000000e+00;
            *((real * )((char *) sv + pitch * 26) + threadID) = 9.343710e-01;
            *((real * )((char *) sv + pitch * 27) + threadID) = 1.000000e+00;
            *((real * )((char *) sv + pitch * 28) + threadID) = 9.998810e-01;
            *((real * )((char *) sv + pitch * 29) + threadID) = 9.999820e-01;
            *((real * )((char *) sv + pitch * 30) + threadID) = 1.000000e+00;
            *((real * )((char *) sv + pitch * 31) + threadID) = 1.000000e+00;
            *((real * )((char *) sv + pitch * 32) + threadID) = 4.830000e-04;
            *((real * )((char *) sv + pitch * 33) + threadID) = 8.180000e-04;
            *((real * )((char *) sv + pitch * 34) + threadID) = 9.983340e-01;
            *((real * )((char *) sv + pitch * 35) + threadID) = 7.600000e-04;
            *((real * )((char *) sv + pitch * 36) + threadID) = 6.260000e-04;
            *((real * )((char *) sv + pitch * 37) + threadID) = 9.000000e-06;
            *((real * )((char *) sv + pitch * 38) + threadID) = 2.720000e-04;
            *((real * )((char *) sv + pitch * 39) + threadID) = 2.568150e-01;
            *((real * )((char *) sv + pitch * 40) + threadID) = 1.480000e-04;
            *((real * )((char *) sv + pitch * 41) + threadID) = 0.000000e+00;
            *((real * )((char *) sv + pitch * 42) + threadID) = 0.000000e+00;
            *((real * )((char *) sv + pitch * 43) + threadID) = 2.978204e+01;
            *((real * )((char *) sv + pitch * 44) + threadID) = 2.978201e+01;
        }
            
        if(use_adpt_dt) {
            *((real *)((char *)sv + pitch * 45) + threadID) = min_dt; // dt
            *((real *)((char *)sv + pitch * 46) + threadID) = 0.0;    // time_new
            *((real *)((char *)sv + pitch * 47) + threadID) = 0.0;    // previous dt
        }
    }
}

__global__ void kernel_set_model_initial_conditions_endo_mid_epi(real *sv, int num_volumes, size_t pitch, bool use_adpt_dt, real min_dt,\
                                                real *initial_endo, real *initial_epi, real *initial_mid, real *transmurality) {
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;

    if (threadID < num_volumes) {
        for (int i = 0; i < NEQ; i++) {
            if (transmurality[threadID] == ENDO)
                *((real * )((char *) sv + pitch * i) + threadID) = initial_endo[i];
            else if (transmurality[threadID] == EPI)
                *((real * )((char *) sv + pitch * i) + threadID) = initial_epi[i];
            else
                *((real * )((char *) sv + pitch * i) + threadID) = initial_mid[i];
        }
            
        if(use_adpt_dt) {
            *((real *)((char *)sv + pitch * 45) + threadID) = min_dt; // dt
            *((real *)((char *)sv + pitch * 46) + threadID) = 0.0;    // time_new
            *((real *)((char *)sv + pitch * 47) + threadID) = 0.0;    // previous dt
        }
    }
}

extern "C" SET_ODE_INITIAL_CONDITIONS_GPU(set_model_initial_conditions_gpu) {

    size_t pitch_h;

    uint8_t use_adpt_dt = (uint8_t)solver->adaptive;

    log_info("Using GPU model implemented in %s\n", __FILE__);

    uint32_t num_volumes = solver->original_num_cells;

    if(use_adpt_dt) {
        log_info("Using Adaptive timestep to solve the ODEs\n");
    } else {
        log_info("Using Fixed timestep to solve the ODEs\n");
    }

    // execution configuration
    const int GRID = (num_volumes + BLOCK_SIZE - 1) / BLOCK_SIZE;

    size_t size = num_volumes * sizeof(real);

    if(use_adpt_dt)
        check_cuda_error(hipMallocPitch((void **)&(solver->sv), &pitch_h, size, (size_t)NEQ + 3));
    else
        check_cuda_error(hipMallocPitch((void **)&(solver->sv), &pitch_h, size, (size_t)NEQ));

    // Get initial condition from extra_data
    real *initial_conditions_endo = NULL;
    real *initial_conditions_epi = NULL;
    real *initial_conditions_mid = NULL;
    real *transmurality = NULL;
    real *initial_conditions_endo_device = NULL;
    real *initial_conditions_epi_device = NULL;
    real *initial_conditions_mid_device = NULL;
    real *transmurality_device = NULL;

    if(solver->ode_extra_data) {
        struct extra_data_for_torord *extra_data = (struct extra_data_for_torord*)solver->ode_extra_data;
        initial_conditions_endo = extra_data->initial_ss_endo;
        initial_conditions_epi = extra_data->initial_ss_epi;
        initial_conditions_mid = extra_data->initial_ss_mid;
        transmurality = extra_data->transmurality;
        check_cuda_error(hipMalloc((void **)&initial_conditions_endo_device, sizeof(real)*NEQ));
        check_cuda_error(hipMemcpy(initial_conditions_endo_device, initial_conditions_endo, sizeof(real)*NEQ, hipMemcpyHostToDevice));
        check_cuda_error(hipMalloc((void **)&initial_conditions_epi_device, sizeof(real)*NEQ));
        check_cuda_error(hipMemcpy(initial_conditions_epi_device, initial_conditions_epi, sizeof(real)*NEQ, hipMemcpyHostToDevice));
        check_cuda_error(hipMalloc((void **)&initial_conditions_mid_device, sizeof(real)*NEQ));
        check_cuda_error(hipMemcpy(initial_conditions_mid_device, initial_conditions_mid, sizeof(real)*NEQ, hipMemcpyHostToDevice));
        check_cuda_error(hipMalloc((void **)&transmurality_device, sizeof(real)*num_volumes));
        check_cuda_error(hipMemcpy(transmurality_device, transmurality, sizeof(real)*num_volumes, hipMemcpyHostToDevice));
    }
    else {
        log_info("[INFO] You should supply a mask function to tag the cells when using this mixed model!\n");
        log_info("[INFO] Considering all cells ENDO!\n");
    }

    if (solver->ode_extra_data) {
        kernel_set_model_initial_conditions_endo_mid_epi<<<GRID, BLOCK_SIZE>>>(solver->sv, num_volumes, pitch_h, use_adpt_dt, solver->min_dt,\
                                                            initial_conditions_endo_device, initial_conditions_epi_device, initial_conditions_mid_device,\
                                                            transmurality_device);
    }
    else {
        kernel_set_model_initial_conditions<<<GRID, BLOCK_SIZE>>>(solver->sv, num_volumes, pitch_h, use_adpt_dt, solver->min_dt);
    }
    

    check_cuda_error(hipPeekAtLastError());
    hipDeviceSynchronize();

    check_cuda_error(hipFree(initial_conditions_endo_device));
    check_cuda_error(hipFree(initial_conditions_epi_device));
    check_cuda_error(hipFree(initial_conditions_mid_device));
    check_cuda_error(hipFree(transmurality_device));

    return pitch_h;
}

extern "C" SOLVE_MODEL_ODES(solve_model_odes_gpu) {

    size_t num_cells_to_solve = ode_solver->num_cells_to_solve;
    uint32_t * cells_to_solve = ode_solver->cells_to_solve;
    real *sv = ode_solver->sv;
    real dt = ode_solver->min_dt;
    uint32_t num_steps = ode_solver->num_steps;

    // execution configuration
    const int GRID = ((int)num_cells_to_solve + BLOCK_SIZE - 1) / BLOCK_SIZE;

    size_t stim_currents_size = sizeof(real) * num_cells_to_solve;
    size_t cells_to_solve_size = sizeof(uint32_t) * num_cells_to_solve;

    real *stims_currents_device = NULL;
    check_cuda_error(hipMalloc((void **)&stims_currents_device, stim_currents_size));
    check_cuda_error(hipMemcpy(stims_currents_device, stim_currents, stim_currents_size, hipMemcpyHostToDevice));

    // the array cells to solve is passed when we are using and adaptive mesh
    uint32_t *cells_to_solve_device = NULL;
    if(cells_to_solve != NULL) {
        check_cuda_error(hipMalloc((void **)&cells_to_solve_device, cells_to_solve_size));
        check_cuda_error(hipMemcpy(cells_to_solve_device, cells_to_solve, cells_to_solve_size, hipMemcpyHostToDevice));
    }

    // Get the extra data array if exists
    uint32_t num_volumes = ode_solver->original_num_cells;
    real *transmurality = NULL;
    real *transmurality_device = NULL;
    int num_extra_parameters = 17;
    real extra_par[num_extra_parameters];
    real *extra_par_device = NULL;
    if(ode_solver->ode_extra_data) {
        struct extra_data_for_torord *extra_data = (struct extra_data_for_torord*)ode_solver->ode_extra_data;
        extra_par[0]  = extra_data->INa_Multiplier; 
        extra_par[1]  = extra_data->ICaL_Multiplier;
        extra_par[2]  = extra_data->Ito_Multiplier;
        extra_par[3]  = extra_data->INaL_Multiplier;
        extra_par[4]  = extra_data->IKr_Multiplier; 
        extra_par[5]  = extra_data->IKs_Multiplier; 
        extra_par[6]  = extra_data->IK1_Multiplier; 
        extra_par[7]  = extra_data->IKb_Multiplier; 
        extra_par[8]  = extra_data->INaCa_Multiplier;
        extra_par[9]  = extra_data->INaK_Multiplier;  
        extra_par[9]  = extra_data->INab_Multiplier;  
        extra_par[10] = extra_data->ICab_Multiplier;  
        extra_par[11] = extra_data->IpCa_Multiplier;  
        extra_par[12] = extra_data->ICaCl_Multiplier;
        extra_par[13] = extra_data->IClb_Multiplier; 
        extra_par[15] = extra_data->Jrel_Multiplier; 
        extra_par[16] = extra_data->Jup_Multiplier;
        transmurality = extra_data->transmurality;
        
        check_cuda_error(hipMalloc((void **)&transmurality_device, sizeof(real)*num_volumes));
        check_cuda_error(hipMemcpy(transmurality_device, transmurality, sizeof(real)*num_volumes, hipMemcpyHostToDevice));

        check_cuda_error(hipMalloc((void **)&extra_par_device, sizeof(real)*num_extra_parameters));
        check_cuda_error(hipMemcpy(extra_par_device, extra_par, sizeof(real)*num_extra_parameters, hipMemcpyHostToDevice));
    }
    else {
        extra_par[0]  = 1.0; 
        extra_par[1]  = 1.0;
        extra_par[2]  = 1.0;
        extra_par[3]  = 1.0;
        extra_par[4]  = 1.0;
        extra_par[5]  = 1.0;
        extra_par[6]  = 1.0; 
        extra_par[7]  = 1.0; 
        extra_par[8]  = 1.0;
        extra_par[9]  = 1.0;
        extra_par[9]  = 1.0; 
        extra_par[10] = 1.0;  
        extra_par[11] = 1.0; 
        extra_par[12] = 1.0;
        extra_par[13] = 1.0;
        extra_par[15] = 1.0;
        extra_par[16] = 1.0;

        check_cuda_error(hipMalloc((void **)&extra_par_device, sizeof(real)*num_extra_parameters));
        check_cuda_error(hipMemcpy(extra_par_device, extra_par, sizeof(real)*num_extra_parameters, hipMemcpyHostToDevice));
    }

    // Transmurality mapping defined on 'extra_data' function
    if (ode_solver->ode_extra_data) {
        solve_endo_mid_epi_gpu<<<GRID, BLOCK_SIZE>>>(current_t, dt, sv, stims_currents_device, cells_to_solve_device, transmurality_device, extra_par_device,\
                                    num_cells_to_solve, num_steps, ode_solver->pitch, ode_solver->adaptive, ode_solver->abs_tol, ode_solver->rel_tol, ode_solver->max_dt);
    }
    // No transmurality: all cells ENDO
    else {
        solve_gpu<<<GRID, BLOCK_SIZE>>>(current_t, dt, sv, stims_currents_device, cells_to_solve_device, extra_par_device,\
                                    num_cells_to_solve, num_steps, ode_solver->pitch, ode_solver->adaptive, ode_solver->abs_tol, ode_solver->rel_tol, ode_solver->max_dt);
    }

    check_cuda_error(hipPeekAtLastError());

    if (stims_currents_device) check_cuda_error(hipFree(stims_currents_device));
    if (cells_to_solve_device) check_cuda_error(hipFree(cells_to_solve_device));
    if (transmurality_device) check_cuda_error(hipFree(transmurality_device));
    if (extra_par_device) check_cuda_error(hipFree(extra_par_device));
}

__global__ void solve_gpu(real cur_time, real dt, real *sv, real *stim_currents, uint32_t *cells_to_solve, real *extra_params,\
                          uint32_t num_cells_to_solve, int num_steps, size_t pitch, bool use_adpt, real abstol, real reltol, real max_dt) {
    const real TOLERANCE = 1e-8;
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int sv_id;

    // Each thread solves one cell model
    if(threadID < num_cells_to_solve) {
        if(cells_to_solve)
            sv_id = cells_to_solve[threadID];
        else
            sv_id = threadID;

        if(!use_adpt) {
            real rDY[NEQ];
            real a[NEQ], b[NEQ];

            for(int n = 0; n < num_steps; ++n) {

                RHS_RL_gpu(a, b, sv, rDY, stim_currents[threadID], 0.0, extra_params, sv_id, dt, pitch, false);

                // Solve variables based on its type:
                //  Non-linear = Euler
                //  Hodkin-Huxley = Rush-Larsen || Euler (if 'a' coefficient is too small)
                SOLVE_EQUATION_EULER_GPU(0);        // v        
                SOLVE_EQUATION_EULER_GPU(1);        // CaMKt    
                SOLVE_EQUATION_EULER_GPU(2);        // cass 
                SOLVE_EQUATION_EULER_GPU(3);        // nai  
                SOLVE_EQUATION_EULER_GPU(4);        // nass 
                SOLVE_EQUATION_EULER_GPU(5);        // ki   
                SOLVE_EQUATION_EULER_GPU(6);        // kss  
                SOLVE_EQUATION_EULER_GPU(7);        // cansr
                SOLVE_EQUATION_EULER_GPU(8);        // cajsr
                SOLVE_EQUATION_EULER_GPU(9);        // cai
                SOLVE_EQUATION_RUSH_LARSEN_GPU(10); // m
                SOLVE_EQUATION_RUSH_LARSEN_GPU(11); // h
                SOLVE_EQUATION_RUSH_LARSEN_GPU(12); // j
                SOLVE_EQUATION_RUSH_LARSEN_GPU(13); // hp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(14); // jp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(15); // mL
                SOLVE_EQUATION_RUSH_LARSEN_GPU(16); // hL
                SOLVE_EQUATION_RUSH_LARSEN_GPU(17); // hLp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(18); // a
                SOLVE_EQUATION_RUSH_LARSEN_GPU(19); // iF
                SOLVE_EQUATION_RUSH_LARSEN_GPU(20); // iS
                SOLVE_EQUATION_RUSH_LARSEN_GPU(21); // ap
                SOLVE_EQUATION_RUSH_LARSEN_GPU(22); // iFp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(23); // iSp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(24); // d
                SOLVE_EQUATION_RUSH_LARSEN_GPU(25); // ff
                SOLVE_EQUATION_RUSH_LARSEN_GPU(26); // fs
                SOLVE_EQUATION_RUSH_LARSEN_GPU(27); // fcaf
                SOLVE_EQUATION_RUSH_LARSEN_GPU(28); // fcas
                SOLVE_EQUATION_RUSH_LARSEN_GPU(29); // jca
                SOLVE_EQUATION_RUSH_LARSEN_GPU(30); // ffp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(31); // fcafp
                SOLVE_EQUATION_EULER_GPU(32);       // nca
                SOLVE_EQUATION_EULER_GPU(33);       // nca_i
                SOLVE_EQUATION_EULER_GPU(34);       // ikr_c0
                SOLVE_EQUATION_EULER_GPU(35);       // ikr_c1
                SOLVE_EQUATION_EULER_GPU(36);       // ikr_c2
                SOLVE_EQUATION_EULER_GPU(37);       // ikr_i
                SOLVE_EQUATION_EULER_GPU(38);       // ikr_o
                SOLVE_EQUATION_RUSH_LARSEN_GPU(39); // xs1
                SOLVE_EQUATION_RUSH_LARSEN_GPU(40); // xs2
                SOLVE_EQUATION_RUSH_LARSEN_GPU(41); // Jrel_np
                SOLVE_EQUATION_RUSH_LARSEN_GPU(42); // Jrel_p
                SOLVE_EQUATION_EULER_GPU(43);       // cli
                SOLVE_EQUATION_EULER_GPU(44);       // clss
            }
        } else {
            //solve_forward_euler_gpu_adpt(sv, stim_currents[threadID], 0.0, extra_params, cur_time + max_dt, sv_id, pitch, abstol,  reltol,  dt,  max_dt);
            solve_rush_larsen_gpu_adpt(sv, stim_currents[threadID], 0.0, extra_params, cur_time + max_dt, sv_id, pitch, abstol,  reltol,  dt,  max_dt);
        }
    }
}

__global__ void solve_endo_mid_epi_gpu(real cur_time, real dt, real *sv, real *stim_currents, uint32_t *cells_to_solve, real *transmurality, real *extra_params,\
                          uint32_t num_cells_to_solve, int num_steps, size_t pitch, bool use_adpt, real abstol, real reltol, real max_dt) {
    const real TOLERANCE = 1e-8;
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int sv_id;

    // Each thread solves one cell model
    if(threadID < num_cells_to_solve) {
        if(cells_to_solve)
            sv_id = cells_to_solve[threadID];
        else
            sv_id = threadID;

        if(!use_adpt) {
            real rDY[NEQ];
            real a[NEQ], b[NEQ];

            for(int n = 0; n < num_steps; ++n) {

                RHS_RL_gpu(a, b, sv, rDY, stim_currents[threadID], transmurality[threadID], extra_params, sv_id, dt, pitch, false);

                // Solve variables based on its type:
                //  Non-linear = Euler
                //  Hodkin-Huxley = Rush-Larsen || Euler (if 'a' coefficient is too small)
                SOLVE_EQUATION_EULER_GPU(0);        // v        
                SOLVE_EQUATION_EULER_GPU(1);        // CaMKt    
                SOLVE_EQUATION_EULER_GPU(2);        // cass 
                SOLVE_EQUATION_EULER_GPU(3);        // nai  
                SOLVE_EQUATION_EULER_GPU(4);        // nass 
                SOLVE_EQUATION_EULER_GPU(5);        // ki   
                SOLVE_EQUATION_EULER_GPU(6);        // kss  
                SOLVE_EQUATION_EULER_GPU(7);        // cansr
                SOLVE_EQUATION_EULER_GPU(8);        // cajsr
                SOLVE_EQUATION_EULER_GPU(9);        // cai
                SOLVE_EQUATION_RUSH_LARSEN_GPU(10); // m
                SOLVE_EQUATION_RUSH_LARSEN_GPU(11); // h
                SOLVE_EQUATION_RUSH_LARSEN_GPU(12); // j
                SOLVE_EQUATION_RUSH_LARSEN_GPU(13); // hp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(14); // jp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(15); // mL
                SOLVE_EQUATION_RUSH_LARSEN_GPU(16); // hL
                SOLVE_EQUATION_RUSH_LARSEN_GPU(17); // hLp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(18); // a
                SOLVE_EQUATION_RUSH_LARSEN_GPU(19); // iF
                SOLVE_EQUATION_RUSH_LARSEN_GPU(20); // iS
                SOLVE_EQUATION_RUSH_LARSEN_GPU(21); // ap
                SOLVE_EQUATION_RUSH_LARSEN_GPU(22); // iFp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(23); // iSp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(24); // d
                SOLVE_EQUATION_RUSH_LARSEN_GPU(25); // ff
                SOLVE_EQUATION_RUSH_LARSEN_GPU(26); // fs
                SOLVE_EQUATION_RUSH_LARSEN_GPU(27); // fcaf
                SOLVE_EQUATION_RUSH_LARSEN_GPU(28); // fcas
                SOLVE_EQUATION_RUSH_LARSEN_GPU(29); // jca
                SOLVE_EQUATION_RUSH_LARSEN_GPU(30); // ffp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(31); // fcafp
                SOLVE_EQUATION_EULER_GPU(32);       // nca
                SOLVE_EQUATION_EULER_GPU(33);       // nca_i
                SOLVE_EQUATION_EULER_GPU(34);       // ikr_c0
                SOLVE_EQUATION_EULER_GPU(35);       // ikr_c1
                SOLVE_EQUATION_EULER_GPU(36);       // ikr_c2
                SOLVE_EQUATION_EULER_GPU(37);       // ikr_i
                SOLVE_EQUATION_EULER_GPU(38);       // ikr_o
                SOLVE_EQUATION_RUSH_LARSEN_GPU(39); // xs1
                SOLVE_EQUATION_RUSH_LARSEN_GPU(40); // xs2
                SOLVE_EQUATION_RUSH_LARSEN_GPU(41); // Jrel_np
                SOLVE_EQUATION_RUSH_LARSEN_GPU(42); // Jrel_p
                SOLVE_EQUATION_EULER_GPU(43);       // cli
                SOLVE_EQUATION_EULER_GPU(44);       // clss
            }
        } else {
            //solve_forward_euler_gpu_adpt(sv, stim_currents[threadID], transmurality[threadID], extra_params, cur_time + max_dt, sv_id, pitch, abstol,  reltol,  dt,  max_dt);
            solve_rush_larsen_gpu_adpt(sv, stim_currents[threadID], transmurality[threadID], extra_params, cur_time + max_dt, sv_id, pitch, abstol,  reltol,  dt,  max_dt);
        }
    }
}

inline __device__ void solve_forward_euler_gpu_adpt(real *sv, real stim_curr, real mapping, real *extra_params, real final_time, int thread_id, size_t pitch, real abstol, real reltol, real min_dt, real max_dt) {

    #define DT *((real *)((char *)sv + pitch * (NEQ)) + thread_id)
    #define TIME_NEW *((real *)((char *)sv + pitch * (NEQ+1)) + thread_id)
    #define PREVIOUS_DT *((real *)((char *)sv + pitch * (NEQ+2)) + thread_id)

    real rDY[NEQ];

    real _tolerances_[NEQ];
    real _aux_tol = 0.0;
    real dt = DT;
    real time_new = TIME_NEW;
    real previous_dt = PREVIOUS_DT;

    real edos_old_aux_[NEQ];
    real edos_new_euler_[NEQ];
    real _k1__[NEQ];
    real _k2__[NEQ];
    real _k_aux__[NEQ];
    real sv_local[NEQ];

    const real _beta_safety_ = 0.8;

    const real __tiny_ = pow(abstol, 2.0);

    if(time_new + dt > final_time) {
        dt = final_time - time_new;
    }

    for(int i = 0; i < NEQ; i++) {
        sv_local[i] = *((real *)((char *)sv + pitch * i) + thread_id);
    }

    RHS_gpu(sv_local, rDY, stim_curr, mapping, extra_params, thread_id, dt, pitch, true);
    time_new += dt;

    for(int i = 0; i < NEQ; i++) {
        _k1__[i] = rDY[i];
    }

	while(1) {

		for(int i = 0; i < NEQ; i++) {
			// stores the old variables in a vector
			edos_old_aux_[i] = sv_local[i];
			// computes euler method
			edos_new_euler_[i] = _k1__[i] * dt + edos_old_aux_[i];
			// steps ahead to compute the rk2 method
			sv_local[i] = edos_new_euler_[i];
		}

		time_new += dt;

		RHS_gpu(sv_local, rDY, stim_curr, mapping, extra_params, thread_id, dt, pitch, true);
		time_new -= dt; // step back

		real greatestError = 0.0, auxError = 0.0;
		
		for(int i = 0; i < NEQ; i++) {

			// stores the new evaluation
			_k2__[i] = rDY[i];
			_aux_tol = fabs(edos_new_euler_[i]) * reltol;
			_tolerances_[i] = (abstol > _aux_tol) ? abstol : _aux_tol;

			// finds the greatest error between  the steps
			auxError = fabs(((dt / 2.0) * (_k1__[i] - _k2__[i])) / _tolerances_[i]);

			greatestError = (auxError > greatestError) ? auxError : greatestError;
		}

		/// adapt the time step
		greatestError += __tiny_;
		previous_dt = dt;

		/// adapt the time step
		dt = _beta_safety_ * dt * sqrt(1.0f / greatestError);

		if(dt < min_dt) {
			dt = min_dt;
		}
		else if(dt > max_dt) {
			dt = max_dt;
		}

		if(time_new + dt > final_time) {
			dt = final_time - time_new;
		}

		// it doesn't accept the solution or accept and risk a NaN
		if(greatestError >= 1.0f && dt > min_dt) {
			// restore the old values to do it again
			for(int i = 0; i < NEQ; i++) {
				sv_local[i] = edos_old_aux_[i];
			}
		
		} else {
			for(int i = 0; i < NEQ; i++) {
				_k_aux__[i] = _k2__[i];
				_k2__[i] = _k1__[i];
				_k1__[i] = _k_aux__[i];
			}

			for(int i = 0; i < NEQ; i++) {
				sv_local[i] = edos_new_euler_[i];
			}

			if(time_new + previous_dt >= final_time) {
				if(final_time == time_new) {
					break;
				} else if(time_new < final_time) {
					dt = previous_dt = final_time - time_new;
					time_new += previous_dt;
					break;
				} 	
			} else {
				time_new += previous_dt;
			}
		}
	}

    for(int i = 0; i < NEQ; i++) {
        *((real *)((char *)sv + pitch * i) + thread_id) = sv_local[i];
    }

    DT = dt;
    TIME_NEW = time_new;
    PREVIOUS_DT = previous_dt;
}

inline __device__ void solve_rush_larsen_gpu_adpt(real *sv, real stim_curr, real mapping, real *extra_params, real final_time, int thread_id, size_t pitch, real abstol, real reltol, real min_dt, real max_dt) {

    #define DT *((real *)((char *)sv + pitch * (NEQ)) + thread_id)
    #define TIME_NEW *((real *)((char *)sv + pitch * (NEQ+1)) + thread_id)
    #define PREVIOUS_DT *((real *)((char *)sv + pitch * (NEQ+2)) + thread_id)

    real rDY[NEQ], a_[NEQ], b_[NEQ], a_new[NEQ], b_new[NEQ];

    real dt = DT;
    real time_new = TIME_NEW;
    real previous_dt = PREVIOUS_DT;

    real edos_old_aux_[NEQ];
    real edos_new_euler_[NEQ];
    real _k1__[NEQ];
    real _k2__[NEQ];
    real _k_aux__[NEQ];
    real sv_local[NEQ];

    const real __tiny_ = pow(abstol, 2.0);

    if(time_new + dt > final_time) {
        dt = final_time - time_new;
    }

    for(int i = 0; i < NEQ; i++) {
        sv_local[i] = *((real *)((char *)sv + pitch * i) + thread_id);
    }

    RHS_RL_gpu(a_, b_, sv_local, rDY, stim_curr, mapping, extra_params, thread_id, dt, pitch, true);
    time_new += dt;

    for(int i = 0; i < NEQ; i++) {
        _k1__[i] = rDY[i];
    }

	while(1) {

        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_EULER_GPU(0);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_EULER_GPU(1);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_EULER_GPU(2);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_EULER_GPU(3);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_EULER_GPU(4);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_EULER_GPU(5);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_EULER_GPU(6);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_EULER_GPU(7);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_EULER_GPU(8);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_EULER_GPU(9);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_RL_GPU(10);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_RL_GPU(11);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_RL_GPU(12);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_RL_GPU(13);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_RL_GPU(14);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_RL_GPU(15);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_RL_GPU(16);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_RL_GPU(17);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_RL_GPU(18);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_RL_GPU(19);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_RL_GPU(20);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_RL_GPU(21);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_RL_GPU(22);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_RL_GPU(23);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_RL_GPU(24);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_RL_GPU(25);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_RL_GPU(26);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_RL_GPU(27);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_RL_GPU(28);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_RL_GPU(29);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_RL_GPU(30);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_RL_GPU(31);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_EULER_GPU(32);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_EULER_GPU(33);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_EULER_GPU(34);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_EULER_GPU(35);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_EULER_GPU(36);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_EULER_GPU(37);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_EULER_GPU(38);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_RL_GPU(39);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_RL_GPU(40);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_RL_GPU(41);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_RL_GPU(42);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_EULER_GPU(43);
        SOLVE_EQUATION_ADAPT_RUSH_LARSEN_EULER_GPU(44);

		time_new += dt;

		RHS_RL_gpu(a_new, b_new, sv_local, rDY, stim_curr, mapping, extra_params, thread_id, dt, pitch, true);
		time_new -= dt; // step back

		real greatestError = 0.0, auxError = 0.0;
		real as, bs, f, y_2nd_order;
		SOLVE_ERROR_ADAPT_RUSH_LARSEN_EULER_GPU(0);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_EULER_GPU(1);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_EULER_GPU(2);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_EULER_GPU(3);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_EULER_GPU(4);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_EULER_GPU(5);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_EULER_GPU(6);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_EULER_GPU(7);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_EULER_GPU(8);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_EULER_GPU(9);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_RL_GPU(10);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_RL_GPU(11);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_RL_GPU(12);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_RL_GPU(13);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_RL_GPU(14);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_RL_GPU(15);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_RL_GPU(16);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_RL_GPU(17);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_RL_GPU(18);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_RL_GPU(19);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_RL_GPU(20);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_RL_GPU(21);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_RL_GPU(22);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_RL_GPU(23);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_RL_GPU(24);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_RL_GPU(25);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_RL_GPU(26);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_RL_GPU(27);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_RL_GPU(28);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_RL_GPU(29);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_RL_GPU(30);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_RL_GPU(31);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_EULER_GPU(32);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_EULER_GPU(33);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_EULER_GPU(34);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_EULER_GPU(35);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_EULER_GPU(36);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_EULER_GPU(37);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_EULER_GPU(38);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_RL_GPU(39);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_RL_GPU(40);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_RL_GPU(41);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_RL_GPU(42);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_EULER_GPU(43);
        SOLVE_ERROR_ADAPT_RUSH_LARSEN_EULER_GPU(44);

		/// adapt the time step
		greatestError += __tiny_;
		previous_dt = dt;

		/// adapt the time step
		dt = dt * sqrt(0.5f * reltol / greatestError);                  

		if(dt < min_dt) {
			dt = min_dt;
		}
		else if(dt > max_dt) {
			dt = max_dt;
		}

		if(time_new + dt > final_time) {
			dt = final_time - time_new;
		}

		// it doesn't accept the solution or accept and risk a NaN
		if(greatestError >= 1.0f && dt > min_dt) {
			// restore the old values to do it again
			for(int i = 0; i < NEQ; i++) {
				sv_local[i] = edos_old_aux_[i];
			}
		
		} else {
			for(int i = 0; i < NEQ; i++) {
				_k_aux__[i] = _k2__[i];
				_k2__[i] = _k1__[i];
				_k1__[i] = _k_aux__[i];

                _k_aux__[i] = a_[i];
                a_[i] = a_new[i];
                a_new[i] = _k_aux__[i];

                _k_aux__[i] = b_[i];
                b_[i] = b_new[i];
                b_new[i] = _k_aux__[i];
			}

			for(int i = 0; i < NEQ; i++) {
				sv_local[i] = edos_new_euler_[i];
			}

			if(time_new + previous_dt >= final_time) {
				if(final_time == time_new) {
					break;
				} else if(time_new < final_time) {
					dt = previous_dt = final_time - time_new;
					time_new += previous_dt;
					break;
				} 	
			} else {
				time_new += previous_dt;
			}
		}
	}

    for(int i = 0; i < NEQ; i++) {
        *((real *)((char *)sv + pitch * i) + thread_id) = sv_local[i];
    }

    DT = dt;
    TIME_NEW = time_new;
    PREVIOUS_DT = previous_dt;
}

inline __device__ void RHS_gpu(real *sv, real *rDY_, real stim_current, real mapping, real *extra_params, int threadID_, real dt, size_t pitch, bool use_adpt_dt) {
    
    // Current modifiers
    real INa_Multiplier   = extra_params[0]; 
    real ICaL_Multiplier  = extra_params[1];
    real Ito_Multiplier   = extra_params[2];
    real INaL_Multiplier  = extra_params[3];
    real IKr_Multiplier   = extra_params[4]; 
    real IKs_Multiplier   = extra_params[5]; 
    real IK1_Multiplier   = extra_params[6]; 
    real IKb_Multiplier   = extra_params[7]; 
    real INaCa_Multiplier = extra_params[8];
    real INaK_Multiplier  = extra_params[9];  
    real INab_Multiplier  = extra_params[10];  
    real ICab_Multiplier  = extra_params[11];  
    real IpCa_Multiplier  = extra_params[12];  
    real ICaCl_Multiplier = extra_params[13];
    real IClb_Multiplier  = extra_params[14]; 
    real Jrel_Multiplier  = extra_params[15]; 
    real Jup_Multiplier   = extra_params[16];

    // Get the celltype for the current cell
    real celltype = mapping;
    
    // Get the stimulus current from the current cell
    real calc_I_stim = stim_current;

    // State variables
    real v;
    real CaMKt;
    real cass;
    real nai;
    real nass;
    real ki;
    real kss;
    real cansr;
    real cajsr;
    real cai;
    real m;
    real h;
    real j;
    real hp;
    real jp;
    real mL;
    real hL;
    real hLp;
    real a;
    real iF;
    real iS;
    real ap;
    real iFp;
    real iSp;
    real d;
    real ff;
    real fs;
    real fcaf;
    real fcas;
    real jca;
    real ffp;
    real fcafp;
    real nca;
    real nca_i;
    real ikr_c0;
    real ikr_c1;
    real ikr_c2;
    real ikr_i;
    real ikr_o;
    real xs1;
    real xs2;
    real Jrel_np;
    real Jrel_p;
    real cli;
    real clss;

    if (use_adpt_dt) {
        v = sv[0];
        CaMKt = sv[1];
        cass = sv[2];
        nai = sv[3];
        nass = sv[4];
        ki = sv[5];
        kss = sv[6];
        cansr = sv[7];
        cajsr = sv[8];
        cai = sv[9];
        m = sv[10];
        h = sv[11];
        j = sv[12];
        hp = sv[13];
        jp = sv[14];
        mL = sv[15];
        hL = sv[16];
        hLp = sv[17];
        a = sv[18];
        iF = sv[19];
        iS = sv[20];
        ap = sv[21];
        iFp = sv[22];
        iSp = sv[23];
        d = sv[24];
        ff = sv[25];
        fs = sv[26];
        fcaf = sv[27];
        fcas = sv[28];
        jca = sv[29];
        ffp = sv[30];
        fcafp = sv[31];
        nca = sv[32];
        nca_i = sv[33];
        ikr_c0 = sv[34];
        ikr_c1 = sv[35];
        ikr_c2 = sv[36];
        ikr_i = sv[37];
        ikr_o = sv[38];
        xs1 = sv[39];
        xs2 = sv[40];
        Jrel_np = sv[41];
        Jrel_p = sv[42];
        cli = sv[43];
        clss = sv[44];
    } else {
        v = *((real *)((char *)sv + pitch * 0) + threadID_);
        CaMKt = *((real *)((char *)sv + pitch * 1) + threadID_);
        cass = *((real *)((char *)sv + pitch * 2) + threadID_);
        nai = *((real *)((char *)sv + pitch * 3) + threadID_);
        nass = *((real *)((char *)sv + pitch * 4) + threadID_);
        ki = *((real *)((char *)sv + pitch * 5) + threadID_);
        kss = *((real *)((char *)sv + pitch * 6) + threadID_);
        cansr = *((real *)((char *)sv + pitch * 7) + threadID_);
        cajsr = *((real *)((char *)sv + pitch * 8) + threadID_);
        cai = *((real *)((char *)sv + pitch * 9) + threadID_);
        m = *((real *)((char *)sv + pitch * 10) + threadID_);
        h = *((real *)((char *)sv + pitch * 11) + threadID_);
        j = *((real *)((char *)sv + pitch * 12) + threadID_);
        hp = *((real *)((char *)sv + pitch * 13) + threadID_);
        jp = *((real *)((char *)sv + pitch * 14) + threadID_);
        mL = *((real *)((char *)sv + pitch * 15) + threadID_);
        hL = *((real *)((char *)sv + pitch * 16) + threadID_);
        hLp = *((real *)((char *)sv + pitch * 17) + threadID_);
        a = *((real *)((char *)sv + pitch * 18) + threadID_);
        iF = *((real *)((char *)sv + pitch * 19) + threadID_);
        iS = *((real *)((char *)sv + pitch * 20) + threadID_);
        ap = *((real *)((char *)sv + pitch * 21) + threadID_);
        iFp = *((real *)((char *)sv + pitch * 22) + threadID_);
        iSp = *((real *)((char *)sv + pitch * 23) + threadID_);
        d = *((real *)((char *)sv + pitch * 24) + threadID_);
        ff = *((real *)((char *)sv + pitch * 25) + threadID_);
        fs = *((real *)((char *)sv + pitch * 26) + threadID_);
        fcaf = *((real *)((char *)sv + pitch * 27) + threadID_);
        fcas = *((real *)((char *)sv + pitch * 28) + threadID_);
        jca = *((real *)((char *)sv + pitch * 29) + threadID_);
        ffp = *((real *)((char *)sv + pitch * 30) + threadID_);
        fcafp = *((real *)((char *)sv + pitch * 31) + threadID_);
        nca = *((real *)((char *)sv + pitch * 32) + threadID_);
        nca_i = *((real *)((char *)sv + pitch * 33) + threadID_);
        ikr_c0 = *((real *)((char *)sv + pitch * 34) + threadID_);
        ikr_c1 = *((real *)((char *)sv + pitch * 35) + threadID_);
        ikr_c2 = *((real *)((char *)sv + pitch * 36) + threadID_);
        ikr_i = *((real *)((char *)sv + pitch * 37) + threadID_);
        ikr_o = *((real *)((char *)sv + pitch * 38) + threadID_);
        xs1 = *((real *)((char *)sv + pitch * 39) + threadID_);
        xs2 = *((real *)((char *)sv + pitch * 40) + threadID_);
        Jrel_np = *((real *)((char *)sv + pitch * 41) + threadID_);
        Jrel_p = *((real *)((char *)sv + pitch * 42) + threadID_);
        cli = *((real *)((char *)sv + pitch * 43) + threadID_);
        clss = *((real *)((char *)sv + pitch * 44) + threadID_);
    }

    #include "ToRORd_dynCl_mixed_endo_mid_epi.common.c"
}

inline __device__ void RHS_RL_gpu(real *a_, real *b_, real *sv, real *rDY_, real stim_current, real mapping, real *extra_params, int threadID_, real dt, size_t pitch, bool use_adpt_dt) {
    
    // Current modifiers
    real INa_Multiplier   = extra_params[0]; 
    real ICaL_Multiplier  = extra_params[1];
    real Ito_Multiplier   = extra_params[2];
    real INaL_Multiplier  = extra_params[3];
    real IKr_Multiplier   = extra_params[4]; 
    real IKs_Multiplier   = extra_params[5]; 
    real IK1_Multiplier   = extra_params[6]; 
    real IKb_Multiplier   = extra_params[7]; 
    real INaCa_Multiplier = extra_params[8];
    real INaK_Multiplier  = extra_params[9];  
    real INab_Multiplier  = extra_params[10];  
    real ICab_Multiplier  = extra_params[11];  
    real IpCa_Multiplier  = extra_params[12];  
    real ICaCl_Multiplier = extra_params[13];
    real IClb_Multiplier  = extra_params[14]; 
    real Jrel_Multiplier  = extra_params[15]; 
    real Jup_Multiplier   = extra_params[16];

    // Get the celltype for the current cell
    real celltype = mapping;
    
    // Get the stimulus current from the current cell
    real calc_I_stim = stim_current;

    // State variables
    real v;
    real CaMKt;
    real cass;
    real nai;
    real nass;
    real ki;
    real kss;
    real cansr;
    real cajsr;
    real cai;
    real m;
    real h;
    real j;
    real hp;
    real jp;
    real mL;
    real hL;
    real hLp;
    real a;
    real iF;
    real iS;
    real ap;
    real iFp;
    real iSp;
    real d;
    real ff;
    real fs;
    real fcaf;
    real fcas;
    real jca;
    real ffp;
    real fcafp;
    real nca;
    real nca_i;
    real ikr_c0;
    real ikr_c1;
    real ikr_c2;
    real ikr_i;
    real ikr_o;
    real xs1;
    real xs2;
    real Jrel_np;
    real Jrel_p;
    real cli;
    real clss;

    if (use_adpt_dt) {
        v = sv[0];
        CaMKt = sv[1];
        cass = sv[2];
        nai = sv[3];
        nass = sv[4];
        ki = sv[5];
        kss = sv[6];
        cansr = sv[7];
        cajsr = sv[8];
        cai = sv[9];
        m = sv[10];
        h = sv[11];
        j = sv[12];
        hp = sv[13];
        jp = sv[14];
        mL = sv[15];
        hL = sv[16];
        hLp = sv[17];
        a = sv[18];
        iF = sv[19];
        iS = sv[20];
        ap = sv[21];
        iFp = sv[22];
        iSp = sv[23];
        d = sv[24];
        ff = sv[25];
        fs = sv[26];
        fcaf = sv[27];
        fcas = sv[28];
        jca = sv[29];
        ffp = sv[30];
        fcafp = sv[31];
        nca = sv[32];
        nca_i = sv[33];
        ikr_c0 = sv[34];
        ikr_c1 = sv[35];
        ikr_c2 = sv[36];
        ikr_i = sv[37];
        ikr_o = sv[38];
        xs1 = sv[39];
        xs2 = sv[40];
        Jrel_np = sv[41];
        Jrel_p = sv[42];
        cli = sv[43];
        clss = sv[44];
    } else {
        v = *((real *)((char *)sv + pitch * 0) + threadID_);
        CaMKt = *((real *)((char *)sv + pitch * 1) + threadID_);
        cass = *((real *)((char *)sv + pitch * 2) + threadID_);
        nai = *((real *)((char *)sv + pitch * 3) + threadID_);
        nass = *((real *)((char *)sv + pitch * 4) + threadID_);
        ki = *((real *)((char *)sv + pitch * 5) + threadID_);
        kss = *((real *)((char *)sv + pitch * 6) + threadID_);
        cansr = *((real *)((char *)sv + pitch * 7) + threadID_);
        cajsr = *((real *)((char *)sv + pitch * 8) + threadID_);
        cai = *((real *)((char *)sv + pitch * 9) + threadID_);
        m = *((real *)((char *)sv + pitch * 10) + threadID_);
        h = *((real *)((char *)sv + pitch * 11) + threadID_);
        j = *((real *)((char *)sv + pitch * 12) + threadID_);
        hp = *((real *)((char *)sv + pitch * 13) + threadID_);
        jp = *((real *)((char *)sv + pitch * 14) + threadID_);
        mL = *((real *)((char *)sv + pitch * 15) + threadID_);
        hL = *((real *)((char *)sv + pitch * 16) + threadID_);
        hLp = *((real *)((char *)sv + pitch * 17) + threadID_);
        a = *((real *)((char *)sv + pitch * 18) + threadID_);
        iF = *((real *)((char *)sv + pitch * 19) + threadID_);
        iS = *((real *)((char *)sv + pitch * 20) + threadID_);
        ap = *((real *)((char *)sv + pitch * 21) + threadID_);
        iFp = *((real *)((char *)sv + pitch * 22) + threadID_);
        iSp = *((real *)((char *)sv + pitch * 23) + threadID_);
        d = *((real *)((char *)sv + pitch * 24) + threadID_);
        ff = *((real *)((char *)sv + pitch * 25) + threadID_);
        fs = *((real *)((char *)sv + pitch * 26) + threadID_);
        fcaf = *((real *)((char *)sv + pitch * 27) + threadID_);
        fcas = *((real *)((char *)sv + pitch * 28) + threadID_);
        jca = *((real *)((char *)sv + pitch * 29) + threadID_);
        ffp = *((real *)((char *)sv + pitch * 30) + threadID_);
        fcafp = *((real *)((char *)sv + pitch * 31) + threadID_);
        nca = *((real *)((char *)sv + pitch * 32) + threadID_);
        nca_i = *((real *)((char *)sv + pitch * 33) + threadID_);
        ikr_c0 = *((real *)((char *)sv + pitch * 34) + threadID_);
        ikr_c1 = *((real *)((char *)sv + pitch * 35) + threadID_);
        ikr_c2 = *((real *)((char *)sv + pitch * 36) + threadID_);
        ikr_i = *((real *)((char *)sv + pitch * 37) + threadID_);
        ikr_o = *((real *)((char *)sv + pitch * 38) + threadID_);
        xs1 = *((real *)((char *)sv + pitch * 39) + threadID_);
        xs2 = *((real *)((char *)sv + pitch * 40) + threadID_);
        Jrel_np = *((real *)((char *)sv + pitch * 41) + threadID_);
        Jrel_p = *((real *)((char *)sv + pitch * 42) + threadID_);
        cli = *((real *)((char *)sv + pitch * 43) + threadID_);
        clss = *((real *)((char *)sv + pitch * 44) + threadID_);
    }

    #include "ToRORd_dynCl_mixed_endo_mid_epi_RL.common.c"
}
