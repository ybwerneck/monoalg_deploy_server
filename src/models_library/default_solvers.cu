#include "hip/hip_runtime.h"
extern "C" SET_ODE_INITIAL_CONDITIONS_GPU(set_model_initial_conditions_gpu) {

    size_t pitch_h;

    uint8_t use_adpt_dt = (uint8_t)solver->adaptive;

    //TODO: set a model name??
    log_info("Using GPU model implemented in %s\n", __FILE__);

    uint32_t num_volumes = solver->original_num_cells;

    if(use_adpt_dt) {
        log_info("Using Adaptive Euler model to solve the ODEs\n");
    } else {
        log_info("Using Euler model to solve the ODEs\n");
    }

    // execution configuration
    const int GRID = (num_volumes + BLOCK_SIZE - 1) / BLOCK_SIZE;

    size_t size = num_volumes * sizeof(real);

    if(use_adpt_dt)
        check_cuda_error(hipMallocPitch((void **)&(solver->sv), &pitch_h, size, (size_t)NEQ + 3));
    else
        check_cuda_error(hipMallocPitch((void **)&(solver->sv), &pitch_h, size, (size_t)NEQ));

    kernel_set_model_initial_conditions<<<GRID, BLOCK_SIZE>>>(solver->sv, num_volumes, pitch_h, use_adpt_dt, solver->min_dt);

    check_cuda_error(hipPeekAtLastError());
    hipDeviceSynchronize();

    return pitch_h;
}

extern "C" SOLVE_MODEL_ODES(solve_model_odes_gpu) {

    size_t num_cells_to_solve = ode_solver->num_cells_to_solve;
    uint32_t * cells_to_solve = ode_solver->cells_to_solve;
    real *sv = ode_solver->sv;
    real dt = ode_solver->min_dt;
    uint32_t num_steps = ode_solver->num_steps;

    // execution configuration
    const int GRID = ((int)num_cells_to_solve + BLOCK_SIZE - 1) / BLOCK_SIZE;

    size_t stim_currents_size = sizeof(real) * num_cells_to_solve;
    size_t cells_to_solve_size = sizeof(uint32_t) * num_cells_to_solve;

    real *stims_currents_device;
    check_cuda_error(hipMalloc((void **)&stims_currents_device, stim_currents_size));
    check_cuda_error(hipMemcpy(stims_currents_device, stim_currents, stim_currents_size, hipMemcpyHostToDevice));

    // the array cells to solve is passed when we are using and adaptive mesh
    uint32_t *cells_to_solve_device = NULL;
    if(cells_to_solve != NULL) {
        check_cuda_error(hipMalloc((void **)&cells_to_solve_device, cells_to_solve_size));
        check_cuda_error(
            hipMemcpy(cells_to_solve_device, cells_to_solve, cells_to_solve_size, hipMemcpyHostToDevice));
    }

    solve_gpu<<<GRID, BLOCK_SIZE>>>(current_t, dt, sv, stims_currents_device, cells_to_solve_device, num_cells_to_solve,
                                    num_steps, ode_solver->pitch, ode_solver->adaptive, ode_solver->abs_tol,
                                    ode_solver->rel_tol, ode_solver->max_dt);

    check_cuda_error(hipPeekAtLastError());

    check_cuda_error(hipFree(stims_currents_device));
    if(cells_to_solve_device)
        check_cuda_error(hipFree(cells_to_solve_device));
}


inline __device__ void solve_forward_euler_gpu_adpt(real *sv, real stim_curr, real final_time, int thread_id, size_t pitch, real abstol, real reltol, real min_dt, real max_dt) {

    #define DT *((real *)((char *)sv + pitch * (NEQ)) + thread_id)
    #define TIME_NEW *((real *)((char *)sv + pitch * (NEQ+1)) + thread_id)
    #define PREVIOUS_DT *((real *)((char *)sv + pitch * (NEQ+2)) + thread_id)

    real rDY[NEQ];

    real _tolerances_[NEQ];
    real _aux_tol = 0.0;
    real dt = DT;
    real time_new = TIME_NEW;
    real previous_dt = PREVIOUS_DT;

    real edos_old_aux_[NEQ];
    real edos_new_euler_[NEQ];
    real _k1__[NEQ];
    real _k2__[NEQ];
    real _k_aux__[NEQ];
    real sv_local[NEQ];

    const real _beta_safety_ = 0.8;

    const real __tiny_ = pow(abstol, 2.0);

    if(time_new + dt > final_time) {
        dt = final_time - time_new;
    }

    for(int i = 0; i < NEQ; i++) {
        sv_local[i] = *((real *)((char *)sv + pitch * i) + thread_id);
    }

    RHS_gpu(sv_local, rDY, stim_curr, thread_id, dt, pitch, true);
    time_new += dt;

    for(int i = 0; i < NEQ; i++) {
        _k1__[i] = rDY[i];
    }

	while(1) {

		for(int i = 0; i < NEQ; i++) {
			// stores the old variables in a vector
			edos_old_aux_[i] = sv_local[i];
			// //computes euler method
			edos_new_euler_[i] = _k1__[i] * dt + edos_old_aux_[i];
			// steps ahead to compute the rk2 method
			sv_local[i] = edos_new_euler_[i];
		}

		time_new += dt;

		RHS_gpu(sv_local, rDY, stim_curr, thread_id, dt, pitch, true);
		time_new -= dt; // step back

		real greatestError = 0.0, auxError = 0.0;
		
		for(int i = 0; i < NEQ; i++) {

			// stores the new evaluation
			_k2__[i] = rDY[i];
			_aux_tol = fabs(edos_new_euler_[i]) * reltol;
			_tolerances_[i] = (abstol > _aux_tol) ? abstol : _aux_tol;

			// finds the greatest error between  the steps
			auxError = fabs(((dt / 2.0) * (_k1__[i] - _k2__[i])) / _tolerances_[i]);

			greatestError = (auxError > greatestError) ? auxError : greatestError;
		}

		/// adapt the time step
		greatestError += __tiny_;
		previous_dt = dt;

		/// adapt the time step
		dt = _beta_safety_ * dt * sqrt(1.0f / greatestError);

		if(dt < min_dt) {
			dt = min_dt;
		}
		else if(dt > max_dt) {
			dt = max_dt;
		}

		if(time_new + dt > final_time) {
			dt = final_time - time_new;
		}

		// it doesn't accept the solution or accept and risk a NaN
		if(greatestError >= 1.0f && dt > min_dt) {
			// restore the old values to do it again
			for(int i = 0; i < NEQ; i++) {
				sv_local[i] = edos_old_aux_[i];
			}
		
		} else {
			for(int i = 0; i < NEQ; i++) {
				_k_aux__[i] = _k2__[i];
				_k2__[i] = _k1__[i];
				_k1__[i] = _k_aux__[i];
			}

			for(int i = 0; i < NEQ; i++) {
				sv_local[i] = edos_new_euler_[i];
			}

			if(time_new + previous_dt >= final_time) {
				if(final_time == time_new) {
					break;
				} else if(time_new < final_time) {
					dt = previous_dt = final_time - time_new;
					time_new += previous_dt;
					break;
				} 	
			} else {
				time_new += previous_dt;
			}
		}
	}

    for(int i = 0; i < NEQ; i++) {
        *((real *)((char *)sv + pitch * i) + thread_id) = sv_local[i];
    }

    DT = dt;
    TIME_NEW = time_new;
    PREVIOUS_DT = previous_dt;
}

// Solving the model for each cell in the tissue matrix ni x nj
__global__ void solve_gpu(real cur_time, real dt, real *sv, real *stim_currents, uint32_t *cells_to_solve,
                          uint32_t num_cells_to_solve, int num_steps, size_t pitch, bool use_adpt,
                          real abstol, real reltol, real max_dt) {
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int sv_id;

    // Each thread solves one cell model
    if(threadID < num_cells_to_solve) {
        if(cells_to_solve)
            sv_id = cells_to_solve[threadID];
        else
            sv_id = threadID;

        if(!use_adpt) {
            real rDY[NEQ];

            for(int n = 0; n < num_steps; ++n) {

                RHS_gpu(sv, rDY, stim_currents[threadID], sv_id, dt, pitch, false);

                for(int i = 0; i < NEQ; i++) {
                    *((real *)((char *)sv + pitch * i) + sv_id) =
                        dt * rDY[i] + *((real *)((char *)sv + pitch * i) + sv_id);
                }
            }
        } else {
            solve_forward_euler_gpu_adpt(sv, stim_currents[threadID], cur_time + max_dt, sv_id, pitch, abstol,  reltol,  dt,  max_dt);
        }
    }
}

